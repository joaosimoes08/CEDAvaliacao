#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "fft.h"

// GPU bit-reversal kernel
__global__ void bit_reverse_kernel(hipfftComplex* data, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    int j = 0;
    int temp = i;
    for (int k = 1; k < N; k <<= 1) {
        j <<= 1;
        j |= temp & 1;
        temp >>= 1;
    }
    if (i < j) {
        hipfftComplex temp = data[i];
        data[i] = data[j];
        data[j] = temp;
    }
}

void fft_gpu(ComplexVector& data) {
    int N = data.size();
    hipfftComplex *d_data, *h_data;

    h_data = new hipfftComplex[N];
    for (int i = 0; i < N; i++) {
        h_data[i].x = data[i].real();
        h_data[i].y = data[i].imag();
    }

    hipMalloc((void**)&d_data, N * sizeof(hipfftComplex));
    hipMemcpy(d_data, h_data, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    // Perform FFT
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

    // Apply bit-reversal to match CPU output
    int block_size = 256;
    int grid_size = (N + block_size - 1) / block_size;
    bit_reverse_kernel<<<grid_size, block_size>>>(d_data, N);

    hipMemcpy(h_data, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        data[i] = Complex(h_data[i].x, h_data[i].y);
    }

    hipfftDestroy(plan);
    hipFree(d_data);
    delete[] h_data;
}
