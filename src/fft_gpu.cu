#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "fft.h"

// GPU kernel for bit-reversal permutation (paper Section I-B)
__global__ void bit_reverse_kernel(hipfftComplex* data, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

	// Mirror index calculation (matches CPU implementation)
    int j = 0;
    int temp = i;
    for (int k = 1; k < N; k <<= 1) {
        j <<= 1;
        j |= temp & 1;
        temp >>= 1;
    }
    if (i < j) { // Swap mirrored indices
        hipfftComplex temp = data[i];
        data[i] = data[j];
        data[j] = temp;
    }
}

void fft_gpu(ComplexVector& data) {
    int N = data.size();
    hipfftComplex *d_data, *h_data;

	// Copy data to GPU (paper Section III-B)
    h_data = new hipfftComplex[N];
    for (int i = 0; i < N; i++) {
        h_data[i].x = data[i].real();
        h_data[i].y = data[i].imag();
    }

    hipMalloc((void**)&d_data, N * sizeof(hipfftComplex));
    hipMemcpy(d_data, h_data, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);

    // Execute cuFFT (paper Section III-A)
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

    // Post-processing: Apply bit-reversal to match CPU output (paper Section III-C)
    int block_size = 256;
    int grid_size = (N + block_size - 1) / block_size;
    bit_reverse_kernel<<<grid_size, block_size>>>(d_data, N);

	// Copy results back to CPU
    hipMemcpy(h_data, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
        data[i] = Complex(h_data[i].x, h_data[i].y);
    }
    
	// Cleanup
    hipfftDestroy(plan);
    hipFree(d_data);
    delete[] h_data;
}
