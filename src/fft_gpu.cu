#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "fft.h"

void fft_gpu(ComplexVector& data) {
    int N = data.size();
    hipfftComplex *d_data, *h_data;
    
    // Allocate host memory
    h_data = new hipfftComplex[N];
    
    // Convert input data
    for (int i = 0; i < N; i++) {
        h_data[i].x = data[i].real();
        h_data[i].y = data[i].imag();
    }
    
    // Allocate device memory
    hipMalloc((void**)&d_data, N * sizeof(hipfftComplex));
    hipMemcpy(d_data, h_data, N * sizeof(hipfftComplex), hipMemcpyHostToDevice);
    
    // Create and execute FFT plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    
    // Copy result back
    hipMemcpy(h_data, d_data, N * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    
    // Convert output data
    for (int i = 0; i < N; i++) {
        data[i] = Complex(h_data[i].x, h_data[i].y);
    }
    
    // Cleanup
    hipfftDestroy(plan);
    hipFree(d_data);
    delete[] h_data;
}
